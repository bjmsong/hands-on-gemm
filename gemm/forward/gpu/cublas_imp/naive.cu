#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "../helper.h" 

int main(int argc, char** argv){
    int version;
    hipblasStatus_t status = cublasGetVersion(hipblasHandle_t(), &version);
    if (status == HIPBLAS_STATUS_SUCCESS) {
        std::cout << "cuBLAS version: " << version << std::endl;
    } else {
        std::cerr << "Failed to get cuBLAS version" << std::endl;
        return 1;
    }

    int M = std::atoi(argv[1]);
    int N = std::atoi(argv[2]);
    int K = std::atoi(argv[3]);

    size_t bytes_a = M * N * sizeof(float);
    size_t bytes_b = N * K * sizeof(float);
    size_t bytes_c = M * K * sizeof(float);

    float* h_a = (float*)malloc(bytes_a);
    float* h_b = (float*)malloc(bytes_b);
    float* h_c = (float*)malloc(bytes_c);

    matrix_init(h_a, M, N);
    matrix_init(h_b, N, K);

    float *d_a, *d_b, *d_c;
    checkCuda(hipMalloc(&d_a, bytes_a));
    checkCuda(hipMalloc(&d_b, bytes_b));
    checkCuda(hipMalloc(&d_c, bytes_c));

    checkCuda(hipMemcpy(d_a, h_a, bytes_a, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_b, h_b, bytes_b, hipMemcpyHostToDevice));

    int BLOCK_SIZE = 32;
    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 grid(GRID_SIZE, GRID_SIZE);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    hipblasHandle_t handle;
	hipblasCreate(&handle);
    float alpha = 1.0f;
	float beta = 0.0f;
    int WARMUP_TIMES = 100;
    for (int n_count=0; n_count < WARMUP_TIMES; n_count++){
        // c = (alpha*a) * b + (beta*c)
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, K, M, N,
        &alpha, d_b, K, d_a, N, &beta, d_c, K);
    }

    hipEvent_t start, end;
    checkCuda(hipEventCreate(&start));
    checkCuda(hipEventCreate(&end));
    checkCuda(hipEventRecord(start));

    hipDeviceSynchronize();
    int EXECUTE_TIMES = 100;
    for (int n_count=0; n_count < EXECUTE_TIMES; n_count++){
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, K, M, N, 
        &alpha, d_b, K, d_a, N, &beta, d_c, K);
    }
    hipDeviceSynchronize();

    checkCuda(hipEventRecord(end));
    checkCuda(hipEventSynchronize(start));
    checkCuda(hipEventSynchronize(end));

    float msec;
    hipEventElapsedTime(&msec, start, end);
    printf("spend %f ms with size of (%d, %d, %d)\n", msec/EXECUTE_TIMES, M, N, K);
    printf("Computational Throughput: %f TFLOPS\n", (float)2*M*N*K*1e-9*EXECUTE_TIMES/msec);

    checkCuda(hipMemcpy(h_c, d_c, bytes_c, hipMemcpyDeviceToHost));
    checkResult(d_a, d_b, h_c, bytes_c, M, N, K);

    free(h_a);
    free(h_b);
    free(h_c);

    checkCuda(hipFree(d_a));
    checkCuda(hipFree(d_b));
    checkCuda(hipFree(d_c));

    checkcuBLAS(hipblasDestroy(handle));
}