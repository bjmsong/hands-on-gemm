#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "helper.h" 

int main(int argc, char** argv){
    int version;
    hipblasStatus_t status = cublasGetVersion(hipblasHandle_t(), &version);
    if (status == HIPBLAS_STATUS_SUCCESS) {
        std::cout << "cuBLAS version: " << version << std::endl;
    } else {
        std::cerr << "Failed to get cuBLAS version" << std::endl;
        return 1;
    }

    int N = std::atoi(argv[1]);
    int M = N;
    int K = N;

    size_t bytes_a = M * N * sizeof(half);
    size_t bytes_b = N * K * sizeof(half);
    size_t bytes_c = M * K * sizeof(half);

    half* h_a = (half*)malloc(bytes_a);
    half* h_b = (half*)malloc(bytes_b);
    half* h_c = (half*)malloc(bytes_c);

    matrix_init(h_a, M, N);
    matrix_init(h_b, N, K);

    half *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes_a);
    hipMalloc(&d_b, bytes_b);
    hipMalloc(&d_c, bytes_c);

    hipMemcpy(d_a, h_a, bytes_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes_b, hipMemcpyHostToDevice);

    int BLOCK_SIZE = 32;
    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 grid(GRID_SIZE, GRID_SIZE);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    hipblasHandle_t handle;
	hipblasCreate(&handle);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

	__half alpha = 1.0f;
	__half beta = 0.0f;
    hipDeviceSynchronize();
    int EXECUTE_TIMES = 100;
    for (int n_count=0; n_count<EXECUTE_TIMES; n_count++){
        // c = (alpha*a) * b + (beta*c)
        hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, 
        &alpha, d_b, K, d_a, N, &beta, d_c, K);
    }
    hipDeviceSynchronize();

    hipEventRecord(end);
    hipEventSynchronize(start);
    hipEventSynchronize(end);

    float msec;
    hipEventElapsedTime(&msec, start, end);
    printf("spend %f ms with size of (%d, %d, %d)\n", msec/EXECUTE_TIMES, M, N, K);
    printf("Computational Throughput: %f TFLOPS\n", (float)2*M*N*K*1e-9*EXECUTE_TIMES/msec);

    hipMemcpy(h_c, d_c, bytes_c, hipMemcpyDeviceToHost);
    checkResult(d_a, d_b, h_c, bytes_c, M, N, K);

    free(h_a);
    free(h_b);
    free(h_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}