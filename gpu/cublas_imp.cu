#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "helper.h" 

int main(int argc, char** argv){
    int N = std::atoi(argv[1]);
    int M = N;
    int K = N;

    size_t bytes_a = M * N * sizeof(float);
    size_t bytes_b = N * K * sizeof(float);
    size_t bytes_c = M * K * sizeof(float);

    float* h_a = (float*)malloc(bytes_a);
    float* h_b = (float*)malloc(bytes_b);
    float* h_c = (float*)malloc(bytes_c);

    matrix_init(h_a, M, N);
    matrix_init(h_b, N, K);

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes_a);
    hipMalloc(&d_b, bytes_b);
    hipMalloc(&d_c, bytes_c);

    hipMemcpy(d_a, h_a, bytes_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes_b, hipMemcpyHostToDevice);

    int BLOCK_SIZE = 32;
    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 grid(GRID_SIZE, GRID_SIZE);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    hipblasHandle_t handle;
	hipblasCreate(&handle);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

	float alpha = 1.0f;
	float beta = 0.0f;
    // Calculate: c = (alpha*a) * b + (beta*c)
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, 
    &alpha, d_b, K, d_a, N, &beta, d_c, K);

    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec;
    hipEventElapsedTime(&msec, start, end);
    printf("spend %f ms with size of (%d, %d, %d)\n", msec, M, N, K);
    printf("Computational Throughput: %f TFLOPS\n", (float)2*M*N*K*1e-9/msec);

    hipMemcpy(h_c, d_c, bytes_c, hipMemcpyDeviceToHost);
    checkResult(d_a, d_b, h_c, bytes_c, M, N, K);

    free(h_a);
    free(h_b);
    free(h_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}