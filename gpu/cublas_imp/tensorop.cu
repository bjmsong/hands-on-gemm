#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "../helper.h" 

int main(int argc, char** argv){
    int version;
    hipblasStatus_t status = cublasGetVersion(hipblasHandle_t(), &version);
    if (status == HIPBLAS_STATUS_SUCCESS) {
        std::cout << "cuBLAS version: " << version << std::endl;
    } else {
        std::cerr << "Failed to get cuBLAS version" << std::endl;
        return 1;
    }

    int M = std::atoi(argv[1]);
    int N = std::atoi(argv[2]);
    int K = std::atoi(argv[3]);

    size_t bytes_a = M * N * sizeof(half);
    size_t bytes_b = N * K * sizeof(half);
    size_t bytes_c = M * K * sizeof(float);

    half* h_a = (half*)malloc(bytes_a);
    half* h_b = (half*)malloc(bytes_b);
    float* h_c = (float*)malloc(bytes_c);

    matrix_init(h_a, M, N);
    matrix_init(h_b, N, K);

    half *d_a, *d_b;
    float *d_c;
    checkCuda(hipMalloc(&d_a, bytes_a));
    checkCuda(hipMalloc(&d_b, bytes_b));
    checkCuda(hipMalloc(&d_c, bytes_c));

    checkCuda(hipMemcpy(d_a, h_a, bytes_a, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_b, h_b, bytes_b, hipMemcpyHostToDevice));

    hipblasHandle_t handle;
	hipblasCreate(&handle);
    float alpha = 1.0f;
	float beta = 0.0f;
    int WARMUP_TIMES = 100;
    for (int n_count=0;n_count<WARMUP_TIMES;n_count++){
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, K, M, N, 
        &alpha, d_b, HIP_R_16F, K, d_a, HIP_R_16F, N, &beta, d_c, 
        HIP_R_32F, K, HIP_R_32F, static_cast<hipblasGemmAlgo_t>(HIPBLAS_GEMM_DEFAULT));
    }

    hipEvent_t start, end;
    checkCuda(hipEventCreate(&start));
    checkCuda(hipEventCreate(&end));
    checkCuda(hipEventRecord(start));
    
    hipDeviceSynchronize();
    int EXECUTE_TIMES = 100;
    for (int n_count=0; n_count<EXECUTE_TIMES; n_count++){
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, K, M, N, 
        &alpha, d_b, HIP_R_16F, K, d_a, HIP_R_16F, N, &beta, d_c, 
        HIP_R_32F, K, HIP_R_32F, static_cast<hipblasGemmAlgo_t>(HIPBLAS_GEMM_DEFAULT));
    }
    hipDeviceSynchronize();

    checkCuda(hipEventRecord(end));
    checkCuda(hipEventSynchronize(start));
    checkCuda(hipEventSynchronize(end));

    float msec;
    hipEventElapsedTime(&msec, start, end);
    printf("spend %f ms with size of (%d, %d, %d)\n", msec/EXECUTE_TIMES, M, N, K);
    printf("Computational Throughput: %f TFLOPS\n", (float)2*M*N*K*1e-9*EXECUTE_TIMES/msec);

    checkCuda(hipMemcpy(h_c, d_c, bytes_c, hipMemcpyDeviceToHost));
    checkResult(d_a, d_b, h_c, bytes_c, M, N, K);

    free(h_a);
    free(h_b);
    free(h_c);

    checkCuda(hipFree(d_a));
    checkCuda(hipFree(d_b));
    checkCuda(hipFree(d_c));

    checkcuBLAS(hipblasDestroy(handle));
}