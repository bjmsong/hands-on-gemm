#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../helper.h" 

#define TILE_WIDTH 16 
__global__ void matrixMultipy(float* a, float* b, float* c, int M, int N, int K){

    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // each thread calculate (row, col) of Matrix C
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float temp = 0;
    for(int ph=0; ph<N/TILE_WIDTH; ph++){
        // load by row
        if((row < M) && (ph*TILE_WIDTH + threadIdx.x) < N)
            Mds[threadIdx.y][threadIdx.x] = a[row * N + ph * TILE_WIDTH + threadIdx.x];
        else
            Mds[threadIdx.y][threadIdx.x] = 0.0f;
        // load by col
        if((col < K) && (ph*TILE_WIDTH + threadIdx.y) < N)
            Nds[threadIdx.y][threadIdx.x] = b[(ph*TILE_WIDTH+threadIdx.y)*K + col];
        else
            Nds[threadIdx.y][threadIdx.x] = 0.0f;
        __syncthreads();

        for(int i = 0; i < TILE_WIDTH; i++)
            temp += Mds[threadIdx.y][i] * Nds[i][threadIdx.x];
        __syncthreads();
    }
    if ((row < M) && (col < K)){
        c[row*K + col] = temp;
    }
}

int main(int argc, char** argv){
    int M = std::atoi(argv[1]);
    int N = std::atoi(argv[2]);
    int K = std::atoi(argv[3]);

    size_t bytes_a = M * N * sizeof(float);
    size_t bytes_b = N * K * sizeof(float);
    size_t bytes_c = M * K * sizeof(float);

    float* h_a = (float*)malloc(bytes_a);
    float* h_b = (float*)malloc(bytes_b);
    float* h_c = (float*)malloc(bytes_c);

    matrix_init(h_a, M, N);
    matrix_init(h_b, N, K);

    float *d_a, *d_b, *d_c;
    checkCuda(hipMalloc(&d_a, bytes_a));
    checkCuda(hipMalloc(&d_b, bytes_b));
    checkCuda(hipMalloc(&d_c, bytes_c));

    checkCuda(hipMemcpy(d_a, h_a, bytes_a, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_b, h_b, bytes_b, hipMemcpyHostToDevice));

    int BLOCK_SIZE = TILE_WIDTH;
    int GRID_SIZE_X = (K + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int GRID_SIZE_Y = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 grid(GRID_SIZE_X, GRID_SIZE_Y);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    int WARMUP_TIMES = 100;
    for (int n_count=0; n_count < WARMUP_TIMES; n_count++){
        matrixMultipy<<<grid, block>>>(d_a, d_b, d_c, M, N, K);
    }
    
    hipEvent_t start, end;
    checkCuda(hipEventCreate(&start));
    checkCuda(hipEventCreate(&end));
    checkCuda(hipEventRecord(start));
    hipDeviceSynchronize();
    int EXECUTE_TIMES = 100;
    for (int n_count=0;n_count<EXECUTE_TIMES;n_count++){
        matrixMultipy<<<grid, block>>>(d_a, d_b, d_c, M, N, K);
    }
    hipDeviceSynchronize();
    checkCuda(hipEventRecord(end));
    checkCuda(hipEventSynchronize(start));
    checkCuda(hipEventSynchronize(end));

    float msec;
    hipEventElapsedTime(&msec, start, end);

    checkCuda(hipMemcpy(h_c, d_c, bytes_c, hipMemcpyDeviceToHost));
    // checkResult(d_a, d_b, h_c, bytes_c, M, N, K);

    free(h_a);
    free(h_b);
    free(h_c);

    checkCuda(hipFree(d_a));
    checkCuda(hipFree(d_b));
    checkCuda(hipFree(d_c));

    printf("spend %f ms with size of (%d, %d, %d)\n", msec/EXECUTE_TIMES, M, N, K);
    printf("Computational Throughput: %f TFLOPS\n", (float)2*M*N*K*1e-9*EXECUTE_TIMES/msec);
}