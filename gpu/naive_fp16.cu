#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "helper.h" 

__global__ void matrixMultipy(half* a, half* b, half* c, int M, int N, int K){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float temp = 0;
    if (row < M && col < K){
        for (int i=0; i<N; i++){
            // 转换会带来误差
            temp += __half2float(a[row * N + i]) * __half2float(b[i * K + col]);
        }
        c[row * K + col] = __float2half(temp);
    }
}

int main(int argc, char** argv){
    int N = std::atoi(argv[1]);
    int M = N;
    int K = N;

    size_t bytes_a = M * N * sizeof(half);
    size_t bytes_b = N * K * sizeof(half);
    size_t bytes_c = M * K * sizeof(half);

    half* h_a = (half*)malloc(bytes_a);
    half* h_b = (half*)malloc(bytes_b);
    half* h_c = (half*)malloc(bytes_c);

    matrix_init(h_a, M, N);
    matrix_init(h_b, N, K);

    half *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes_a);
    hipMalloc(&d_b, bytes_b);
    hipMalloc(&d_c, bytes_c);

    hipMemcpy(d_a, h_a, bytes_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes_b, hipMemcpyHostToDevice);

    int BLOCK_SIZE = 16;
    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 grid(GRID_SIZE, GRID_SIZE);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    hipDeviceSynchronize();
    int EXECUTE_TIMES = 100;
    for (int n_count=0;n_count<EXECUTE_TIMES;n_count++){
        matrixMultipy<<<grid, block>>>(d_a, d_b, d_c, M, N, K);
    }
    hipDeviceSynchronize();

    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec;
    hipEventElapsedTime(&msec, start, end);

    hipMemcpy(h_c, d_c, bytes_c, hipMemcpyDeviceToHost);
    checkResult(d_a, d_b, h_c, bytes_c, M, N, K);

    free(h_a);
    free(h_b);
    free(h_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    printf("spend %f ms with size of (%d, %d, %d)\n", msec/EXECUTE_TIMES, M, N, K);
    printf("Computational Throughput: %f TFLOPS\n", (float)2*M*N*K*1e-9*EXECUTE_TIMES/msec);
}