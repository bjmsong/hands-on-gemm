#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "helper.h" 

#define TILE_WIDTH 16 
#define CORSE_FATOR 4
__global__ void matrixMultipy(float*  a, float* b, float* c, int M, int N, int K){

    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // each thread calculate #CORSE_FATOR points of Matrix C
    // (row, colStart), (row, colStart + TILE_WIDTH), ... (row, colStart + (CORSE_FATOR-1)*TILE_WIDTH)
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int colStart = blockIdx.x * blockDim.x * CORSE_FATOR + threadIdx.x;

    float temp[CORSE_FATOR];
    for (int f = 0; f < CORSE_FATOR; f++){
        temp[f] = 0.0f;
    }
    if(row < M && colStart < K){
        for(int ph=0; ph<N/TILE_WIDTH; ph++){
            // load by row
            Mds[threadIdx.y][threadIdx.x] = a[row * N + ph * TILE_WIDTH + threadIdx.x];
            // load by col
            for (int f = 0; f < CORSE_FATOR; f++){
                Nds[threadIdx.y][threadIdx.x] = b[(ph*TILE_WIDTH+threadIdx.y)*K + colStart + f * TILE_WIDTH];
                __syncthreads();

            for(int i = 0; i < TILE_WIDTH; i++)
                temp[f] += Mds[threadIdx.y][i] * Nds[i][threadIdx.x];
            __syncthreads();
        }
        for (int f = 0; f < CORSE_FATOR; f++)
            c[row*K + colStart + f * TILE_WIDTH] = temp[f];
    }
    }
}

int main(int argc, char** argv){
    int N = std::atoi(argv[1]);
    int M = N;
    int K = N;

    size_t bytes_a = M * N * sizeof(float);
    size_t bytes_b = N * K * sizeof(float);
    size_t bytes_c = M * K * sizeof(float);

    float* h_a = (float*)malloc(bytes_a);
    float* h_b = (float*)malloc(bytes_b);
    float* h_c = (float*)malloc(bytes_c);

    matrix_init(h_a, M, N);
    matrix_init(h_b, N, K);

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes_a);
    hipMalloc(&d_b, bytes_b);
    hipMalloc(&d_c, bytes_c);

    hipMemcpy(d_a, h_a, bytes_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes_b, hipMemcpyHostToDevice);

    int BLOCK_SIZE = TILE_WIDTH;
    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 grid(GRID_SIZE/CORSE_FATOR, GRID_SIZE);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    matrixMultipy<<<grid, block>>>(d_a, d_b, d_c, N, N, N);
    
    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec;
    hipEventElapsedTime(&msec, start, end);

    hipMemcpy(h_c, d_c, bytes_c, hipMemcpyDeviceToHost);
    // checkResult(d_a, d_b, h_c, bytes_c, M, N, K);

    free(h_a);
    free(h_b);
    free(h_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    printf("spend %f ms with size of (%d, %d, %d)\n", msec, M, N, K);
    printf("Computational Throughput: %f TFLOPS\n", (float)2*M*N*K*1e-9/msec);
}